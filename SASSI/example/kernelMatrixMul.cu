
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include <math.h>
#define COL 32
#define ROW 32

__global__ void matrixMulKernel( float *devA, float *devB, float *devC, int row, int col, const int k){
        int txID = blockIdx.x * blockDim.x + threadIdx.x;//Col of devC
        int tyID = blockIdx.y * blockDim.y + threadIdx.y;//Row of devC.
	
        if ((txID < col) && (tyID < row))
	{
		float Pvalue = 0;
		for(int w=0; w<k; w++){
			Pvalue += devA[tyID*k+w] * devB[w*k+txID];
        	}
        	devC[tyID*k+txID] = Pvalue;
	}
}

void matrixMultiplication(float *a, float *b, float *c, int row, int col, int k, int blockX, int blockY)
{
	//Setting device memory space.
        int sizeA = row*k*sizeof(float);
        int sizeB = k*col*sizeof(float);
        int sizeC = row*col*sizeof(float);
        float *devA, *devB, *devC;

	//set cudaDevice
	int num_devices, device;
	hipGetDeviceCount(&num_devices);
	printf("Print number of devices: %d\n", num_devices);
	if (num_devices > 1) {
	      int max_multiprocessors = 0, max_device = 0;
	      for (device = 0; device < num_devices; device++) {
	              hipDeviceProp_t properties;
		      hipGetDeviceProperties(&properties, device);
			printf("Device id %d, %s : %d.%d\n", device, properties.name, properties.major, properties.minor);
	              if (max_multiprocessors < properties.multiProcessorCount) {
	                      max_multiprocessors = properties.multiProcessorCount;
	                      max_device = device;
	              }
	      }
		int chosenDevice = max_device;
		printf("Max Device: %d\n", max_device);
		chosenDevice = 0;
		printf("Chosen device: %d\n", chosenDevice);
//	      cudaSetDevice(max_device);
	      hipSetDevice(chosenDevice);
	}
//	cudaSetDevice(2);
        
	//Time variables
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	hipMalloc((void**)&devA, sizeA);
        hipMalloc((void**)&devB, sizeB);
        hipMalloc((void**)&devC, sizeC);
	//Copying [A] and [B] from host memory to device memory.

        hipMemcpy(devA, a, sizeA, hipMemcpyHostToDevice);
        hipMemcpy(devB, b, sizeB, hipMemcpyHostToDevice);

	//Setting execution configuration.
	dim3 dimBlock(blockX, blockY, 1);
	dim3 dimGrid((COL+dimBlock.x-1)/dimBlock.x, (ROW+dimBlock.y-1)/dimBlock.y, 1);
        printf("\tBlock(%d, %d, %d)\n", dimBlock.x, dimBlock.y, dimBlock.z);
        printf("\tGrid(%d, %d, %d)\n", dimGrid.x, dimGrid.y, dimGrid.z);
	//Launching device computation threads.
        matrixMulKernel<<<dimGrid, dimBlock>>>(devA, devB, devC, row, col, k);
	//Transferring [C] from device to host.
        hipMemcpy(c, devC, sizeC, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	//Display time
	hipEventElapsedTime(&time, start, stop);
	printf("\tParallel Job time: %.2f ms", time);
	//Freeing device matrices.
        hipFree(devA); 
	hipFree(devB); 
	hipFree(devC);
}

bool checkResults(float *test, float *c, int row, int col){
        bool b= true;
        for(int i=0; i<row; i++){
                for(int j=0; j<col; j++){
                        if(test[i*col+j] != c[i*col+j]){
                                b=false;
                                printf("test[%d, %d] = %.2f \t c[%d, %d] = %.2f\n", i, j, test[i*col+j], i, j, c[i*col+j]);
                                break;
                        }
                }
        }
        return b;
}

int main(int argC, char** argV)
{
        float *a, *b, *c; //, *test;
	//Setting matrix parameters.
        int row = ROW;
        int col = COL;
        int k = COL;
	//Setting host memory space.
        a = (float *) malloc(row*k*sizeof(float));
        b = (float *) malloc(k*col*sizeof(float));
        c = (float *) malloc(row*col*sizeof(float));
        //test = (float *) malloc(row*col*sizeof(float));

	//Initializing [A] and [B] with random values from 1 to 10.
        for(int i=0; i<row; i++){
                for(int j=0; j<k; j++){
                        a[i*k+j] = rand()%10;
                }
        }
        for(int i=0; i<k; i++){
                for(int j=0; j<col; j++){
                        b[i*col+j] = rand()%10;
                }
        }
	printf("Matrix Multiplication: \nA[%d, %d] * B[%d, %d] = C[%d, %d]\n", row, k, k, col, row, col);

	//Calling stub function to allocate device memory, perform data transfer, and launch kernel.
	int blockX = 32;
	if (argV[1] != NULL)
		blockX = atoi(argV[1]);
	int blockY = 32;
	if (argV[2] != NULL)
		blockY = atoi(argV[2]);
       
	if (!blockX)
		blockX = 32;
	if (!blockY)
		blockY = 32;
	matrixMultiplication(a, b, c, row, col, k, blockX, blockY);
        
}
