#include "hip/hip_runtime.h"

#include <algorithm>
#include <assert.h>
#include <cupti.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>
#include "sassi_intrinsics.h"
#include "sassi_lazyallocator.hpp"
#include "sassi_dictionary.hpp"
#include <sassi/sassi-function.hpp>

// 8Mb of space for CFG information.
#define POOLSIZE (1024 * 1024 * 8)
#define MAX_FN_STR_LEN 64

// Create a memory pool that we can populate on the device and read on the host.
static __managed__ uint8_t sassi_mempool[POOLSIZE];
static __managed__ int     sassi_mempool_cur;
static __managed__ int sassi_total_instrs, sassi_divergence, sassi_convergence;
// A structure to record a basic block.  We will perform a deep copy
// of SASSI's SASSIBasicBlockParams for each basic block.
struct BLOCK {
  int id;
  unsigned long long weight;
  bool isEntry;
  bool isExit;
  int numInstrs;
  int numSuccs;
  int succs[2];
};

// A structure to record a function's CFG.
struct CFG {
  char fnName[MAX_FN_STR_LEN];
  int numBlocks;
  BLOCK *blocks;
};

// A dictionary of CFGs.
static __managed__ sassi::dictionary<int64_t, CFG*> *sassi_cfg;

// A dictionary of basic blocks.
static __managed__ sassi::dictionary<int64_t, BLOCK*> *sassi_cfg_blocks;

///////////////////////////////////////////////////////////////////////////////////
///
///  Allocate data from the UVM mempool.
///
///////////////////////////////////////////////////////////////////////////////////
__device__ void *simple_malloc(size_t sz)
{
  int ptr = atomicAdd(&sassi_mempool_cur, sz);
  assert ((ptr + sz) <= POOLSIZE);
  return (void*) &(sassi_mempool[ptr]);
}

///////////////////////////////////////////////////////////////////////////////////
///
///  A simple string copy to copy from device memory to our UVM malloc'd region.
///
///////////////////////////////////////////////////////////////////////////////////
__device__ void simple_strncpy(char *dest, const char *src)
{
  int i;
  for (i = 0; i < MAX_FN_STR_LEN-1; i++) {
    char c = src[i];
    if (c == 0) break;
    dest[i] = c;
  }
  dest[i] = '\0';
}

///////////////////////////////////////////////////////////////////////////////////
///
///  A call to this function will be inserted at the beginning of every 
///  CUDA function or kernel.  We will essentially perform a deep copy of the
///  CFG SASSI presents.  All of the copied data only contains static information
///  about the CFG.  In the sassi_basic_block_entry handler, below, we will 
///  record the dynamic number of times the basic block is invoked.
///
///////////////////////////////////////////////////////////////////////////////////
__device__ void sassi_function_entry(SASSIFunctionParams* fp)
{
  int numBlocks = fp->GetNumBlocks();
  const SASSIBasicBlockParams * const * blocks = fp->GetBlocks();
  
  CFG *cPtr = *(sassi_cfg->getOrInit((int64_t)fp, [numBlocks, blocks, fp](CFG **cfg) {
      CFG *cPtr = (CFG*) simple_malloc(sizeof(CFG));
      simple_strncpy(cPtr->fnName, fp->GetFnName());
      cPtr->numBlocks = numBlocks;
      cPtr->blocks = (BLOCK*) simple_malloc(sizeof(BLOCK) * numBlocks);
      *cfg = cPtr;
  }));

  __threadfence();

  for (int bb = 0; bb < numBlocks; bb++) {
    const SASSIBasicBlockParams *blockParam = blocks[bb];
    BLOCK *blockPtr = &(cPtr->blocks[bb]);    
    sassi_cfg_blocks->getOrInit((int64_t)blockParam, [blockParam, blockPtr](BLOCK **bpp) {
	*bpp = blockPtr;
	blockPtr->id = blockParam->GetID();
	blockPtr->weight = 0;
	blockPtr->isEntry = blockParam->IsEntryBlock(); 
	blockPtr->isExit = blockParam->IsExitBlock(); 
	blockPtr->numInstrs = blockParam->GetNumInstrs(); 
	sassi_total_instrs += blockParam->GetNumInstrs();
	blockPtr->numSuccs = blockParam->GetNumSuccs();
 	printf("NumSuccs: %d\n", blockParam->GetNumSuccs());
	blockParam->GetNumSuccs() == 1 ? sassi_convergence++: (blockParam->GetNumSuccs() >=2 ? sassi_divergence++: sassi_divergence += 0);
	assert(blockParam->GetNumSuccs() <= 2);
	const SASSIBasicBlockParams * const * succs = blockParam->GetSuccs();
	for (int s = 0; s < blockParam->GetNumSuccs(); s++) {
	  blockPtr->succs[s] = succs[s]->GetID();
	}
      });
  }
}

///////////////////////////////////////////////////////////////////////////////////
///
///  Simply lookup the basic block in our dictionary, get its "weight" feild
///  and increment it.
///
///////////////////////////////////////////////////////////////////////////////////
__device__ void sassi_basic_block_entry(SASSIBasicBlockParams *bb)
{
  BLOCK **blockStr = sassi_cfg_blocks->getOrInit((int64_t)bb, [](BLOCK **bpp) { assert(0); });
  atomicAdd(&((*blockStr)->weight), 1);
}

///////////////////////////////////////////////////////////////////////////////////
///
///  Print the graph out in "dot" format.  
///  E.g., use:
///
///       dot -Tps -o graph.ps sassi-cfg.dot 
///
///  to render the graph in postscript.
///
///////////////////////////////////////////////////////////////////////////////////
static void sassi_finalize(sassi::lazy_allocator::device_reset_reason unused)
{
  hipDeviceSynchronize();
  printf("Total instructions: %d\n", sassi_total_instrs);
  //printf("numBlocks: %d\n", numBlocks);
  printf("Total number of divergence: %d, and convergences: %d\n ", sassi_divergence, sassi_convergence);
  FILE *cfgFile = fopen("sassi-cfg.dot", "w");
  sassi_cfg->map([cfgFile](int64_t k, CFG* &cfg) {
      fprintf(cfgFile, "digraph %s {\n", cfg->fnName);
      double weightMax = 0.0;
      for (int bb = 0; bb < cfg->numBlocks; bb++) {
	BLOCK *block = &(cfg->blocks[bb]);
	weightMax = std::max(weightMax, (double)block->weight);
      }
      for (int bb = 0; bb < cfg->numBlocks; bb++) {
	BLOCK *block = &(cfg->blocks[bb]);
	int per = block->isExit ? 3 : 1;
	int boxWeight = 100 - std::round(100.0 * ((double)block->weight / weightMax));
	int fontWeight = boxWeight > 40 ? 0 : 100;
	fprintf(cfgFile, "\tBB%d [style=filled,fontcolor=gray%d,shape=box,"
		"peripheries=%d,color=gray%d,label=\"BB%d : %d ins\"];\n", 
		block->id, fontWeight, per, boxWeight, block->id, block->numInstrs);
      }
      for (int bb = 0; bb < cfg->numBlocks; bb++) {
	BLOCK *block = &(cfg->blocks[bb]);
	for (int s = 0; s < block->numSuccs; s++) {
	  fprintf(cfgFile, "\tBB%d -> BB%d;\n", block->id, block->succs[s]);
	}
      }
      fprintf(cfgFile, "}\n");
    });
  fclose(cfgFile);
}

///////////////////////////////////////////////////////////////////////////////////
///
///  Initialize the UVM memory pool and our two dictionaries.  
///
///////////////////////////////////////////////////////////////////////////////////
static void sassi_init()
{
  sassi_mempool_cur = 0;
  sassi_total_instrs = 0;
  sassi_divergence = 0;
  sassi_convergence = 0;
  bzero(sassi_mempool, sizeof(sassi_mempool));
  sassi_cfg = new sassi::dictionary<int64_t, CFG*>(601);
  sassi_cfg_blocks = new sassi::dictionary<int64_t, BLOCK*>(7919);
}


///////////////////////////////////////////////////////////////////////////////////
///
///  
///
///////////////////////////////////////////////////////////////////////////////////
static sassi::lazy_allocator mapAllocator(sassi_init, sassi_finalize);
